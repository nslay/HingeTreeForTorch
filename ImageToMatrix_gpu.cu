#include "hip/hip_runtime.h"
/*-
 * Copyright (c) 2020 Nathan Lay (enslay@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR(S) ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR(S) BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ImageToMatrix.h"

// From: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
// And from: https://stackoverflow.com/questions/39274472/error-function-atomicadddouble-double-has-already-been-defined
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

//#if __CUDA_ARCH__ < 600
#else
static inline __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

namespace bleak {

namespace {

// Can be passed by value between CPU and GPU
template<unsigned int Dimension>
struct Size {
  int64_t data[Dimension];
};

template<unsigned int Dimension>
class RasterCurveGPU {
public:
  typedef Size<Dimension> SizeType;
  typedef SizeType CoordType;

  __device__ RasterCurveGPU(const SizeType &stSize)
  : m_stSize(stSize) { }

  __device__ RasterCurveGPU(const int64_t a_i64Size[Dimension]) {
    for (unsigned int d = 0; d < Dimension; ++d)
      m_stSize.data[d] = a_i64Size[d];
  }

  __device__ int64_t Count() const {
    int64_t count = m_stSize.data[0];

    for (unsigned int d = 1; d < Dimension; ++d)
      count *= m_stSize.data[d];

    return count;
  }

  __device__ const SizeType & GetSize() const { return m_stSize; }

  __device__ int64_t Index(const CoordType &stCoord) const {
    int64_t index = stCoord.data[0];

    for (unsigned int d = 1; d < Dimension; ++d)
      index = m_stSize.data[d] * index + stCoord.data[d];

    return index;
  }

  __device__ int64_t IndexChecked(const CoordType &stCoord) const {
    if (stCoord.data[0] < 0 || stCoord.data[0] >= m_stSize.data[0])
      return -1;

    int64_t index = stCoord.data[0];

    for (unsigned int d = 1; d < Dimension; ++d) {
      if (stCoord.data[d] < 0 || stCoord.data[d] >= m_stSize.data[d])
        return -1;

      index = m_stSize.data[d] * index + stCoord.data[d];
    }

    return index;
  }

  __device__ CoordType Coordinate(int64_t index) const {
    CoordType stCoord;

    for (unsigned int d = Dimension-1; d > 0; --d) {
      const int64_t q = index / m_stSize.data[d];
      const int64_t r = index - q * m_stSize.data[d];
      stCoord.data[d] = r;
      index = q;
    }

    stCoord.data[0] = index;

    return stCoord;
  }

private:
  SizeType m_stSize;
};

template<typename RealType>
__global__ void ExtractMatrixHelper(RealType *d_matrix, const RealType *d_image, const int64_t *d_indexMatrix, int64_t i64Rows, int64_t i64Cols, RealType padValue) {
  const int64_t i = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t j = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;

  if (i < i64Rows && j < i64Cols) {
    const int64_t index = d_indexMatrix[i64Cols*i + j];
    d_matrix[i64Cols*i + j] = (index < 0) ? padValue : d_image[index];
  }
}

template<typename RealType>
__global__ void MapAndAddHelper(RealType *d_diff, int64_t i64Stride, const RealType *d_matrix, const int64_t *d_indexMatrix, int64_t i64Rows, int64_t i64Cols) {
  const int64_t i = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t j = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;

  if (i < i64Rows && j < i64Cols) {
    const int64_t index = d_indexMatrix[i64Cols*i + j];
    if (index >= 0) {
      atomicAdd(d_diff + index*i64Stride, d_matrix[i64Cols*i + j]);
      //d_diff[index*iStride] += d_matrix[iCols*i + j];
    }
  }
}

template<unsigned int Dimension>
__global__ void ExtractIndexMatrixHelper(int64_t *d_matrix, Size<Dimension> stKernelSize, Size<Dimension> stStride, Size<Dimension> stPadding, Size<Dimension> stDilate, Size<Dimension> stOutSize, Size<Dimension+1> stImageSize) {
  typedef RasterCurveGPU<Dimension> RasterType;
  typedef typename RasterType::CoordType CoordType; 

  const int64_t c = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t j = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t i = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;

  RasterType clOutRaster(stOutSize);
  RasterType clKernRaster(stKernelSize);
  RasterType clImageRaster(stImageSize.data+1);

  const int64_t i64Channels = stImageSize.data[0];
  const int64_t i64KernelCount = clKernRaster.Count();  
  const int64_t i64OutCount = clOutRaster.Count();

  if (c < i64Channels && j < i64KernelCount && i < i64OutCount) {
    const int64_t i64InCount = clImageRaster.Count();
    const int64_t indexOffset = c*i64InCount;
    const int64_t jOffset = c*i64KernelCount;
    const int64_t i64Cols = i64Channels * i64KernelCount;

    CoordType stWinCoord = clKernRaster.Coordinate(j);
    CoordType stCoord = clOutRaster.Coordinate(i);

    for (unsigned int d = 0; d < Dimension; ++d)
      stCoord.data[d] = stCoord.data[d] * stStride.data[d] + stWinCoord.data[d] * stDilate.data[d] - stPadding.data[d];

    const int64_t index = clImageRaster.IndexChecked(stCoord);
    d_matrix[i64Cols*i + (j + jOffset)] = (index < 0) ? index : index + indexOffset;
  }
}

} // end anonymous namespace

template<typename RealType, unsigned int Dimension>
void ImageToMatrixBase<RealType, Dimension>::ExtractMatrixGPU(RealType *d_matrix, const RealType *d_image, const int64_t *d_indexMatrix, const int64_t a_i64ImageSize[Dimension+1]) const {
  int64_t i64Rows = 0;
  int64_t i64Cols = 0;
  ComputeMatrixDimensions(i64Rows, i64Cols, a_i64ImageSize);

  const dim3 threadsPerBlock(16,16);
  const dim3 numBlocks((i64Rows + threadsPerBlock.x-1) / threadsPerBlock.x, (i64Cols + threadsPerBlock.y-1) / threadsPerBlock.y);
  ExtractMatrixHelper<<<numBlocks, threadsPerBlock>>>(d_matrix, d_image, d_indexMatrix, i64Rows, i64Cols, padValue);
}

template<typename RealType, unsigned int Dimension>
void ImageToMatrixBase<RealType, Dimension>::MapAndAddGPU(RealType *d_diff, int64_t i64Stride, const RealType *d_matrix, const int64_t *d_indexMatrix, const int64_t a_i64ImageSize[Dimension+1]) const {
  int64_t i64Rows = 0;
  int64_t i64Cols = 0;
  ComputeMatrixDimensions(i64Rows, i64Cols, a_i64ImageSize);

  const dim3 threadsPerBlock(16,16);
  const dim3 numBlocks((i64Rows + threadsPerBlock.x-1) / threadsPerBlock.x, (i64Cols + threadsPerBlock.y-1) / threadsPerBlock.y);
  MapAndAddHelper<<<numBlocks, threadsPerBlock>>>(d_diff, i64Stride, d_matrix, d_indexMatrix, i64Rows, i64Cols);
}

template<typename RealType, unsigned int Dimension>
void ImageToMatrix<RealType, Dimension>::ExtractIndexMatrixGPU(int64_t *d_matrix, const int64_t a_i64ImageSize[Dimension+1]) const {
  const int64_t i64Channels = a_i64ImageSize[0];
  const int64_t i64KernelCount = ComputeKernelCount();
  const int64_t i64OutCount = ComputeOutputCount(a_i64ImageSize);

  const dim3 threadsPerBlock(4,16,8);
  const dim3 numBlocks((i64Channels + threadsPerBlock.x-1) / threadsPerBlock.x, (i64KernelCount + threadsPerBlock.y-1) / threadsPerBlock.y, (i64OutCount + threadsPerBlock.z-1) / threadsPerBlock.z);

  Size<Dimension> stKernelSize;
  Size<Dimension> stStride;
  Size<Dimension> stPadding;
  Size<Dimension> stDilate;
  Size<Dimension> stOutSize;
  Size<Dimension+1> stImageSize;

  std::copy_n(kernelSize.data(), Dimension, stKernelSize.data);
  std::copy_n(stride.data(), Dimension, stStride.data);
  std::copy_n(padding.data(), Dimension, stPadding.data);
  std::copy_n(dilate.data(), Dimension, stDilate.data);
  std::copy_n(ComputeOutputSize(a_i64ImageSize).data(), Dimension, stOutSize.data);
  std::copy_n(a_i64ImageSize, Dimension+1, stImageSize.data);

  ExtractIndexMatrixHelper<<<numBlocks, threadsPerBlock>>>(d_matrix, stKernelSize, stStride, stPadding, stDilate, stOutSize, stImageSize);
}

// Instantiate these functions by instantiating duplicate ImageToMatrixBase
template class ImageToMatrixBase<float, 1>;
template class ImageToMatrixBase<float, 2>;
template class ImageToMatrixBase<float, 3>;

template class ImageToMatrixBase<double, 1>;
template class ImageToMatrixBase<double, 2>;
template class ImageToMatrixBase<double, 3>;

// Instantiate these functions by instantiating duplicate ImageToMatrix
template class ImageToMatrix<float, 1>;
template class ImageToMatrix<float, 2>;
template class ImageToMatrix<float, 3>;

template class ImageToMatrix<double, 1>;
template class ImageToMatrix<double, 2>;
template class ImageToMatrix<double, 3>;

} // end namespace bleak

#if 0
// Test code...
#include <iostream>
#include <vector>

int main(int argc, char **argv) {
  bleak::ImageToMatrix<float, 2> clIm2Col;

  //
  // Reference 2D access pattern for a 6x5 image with padding
  //
  // -1 -1 -1 -1 -1 -1 -1
  // -1  0  1  2  3  4 -1
  // -1  5  6  7  8  9 -1
  // -1 10 11 12 13 14 -1
  // -1 15 16 17 18 19 -1
  // -1 20 21 22 23 24 -1
  // -1 25 26 27 28 29 -1
  // -1 -1 -1 -1 -1 -1 -1
  //

  clIm2Col.kernelSize[0] = 3;
  clIm2Col.kernelSize[1] = 4;

  clIm2Col.stride[0] = 1;
  clIm2Col.stride[1] = 1;

  clIm2Col.dilate[0] = 1;
  clIm2Col.dilate[1] = 1;

  clIm2Col.padding[0] = 1;
  clIm2Col.padding[1] = 1;

  const int64_t a_i64ImageSize[3] = { 3, 6, 5 }; // 3 channels, 6 rows, 5 columns

  if (!clIm2Col.Good(a_i64ImageSize)) {
    std::cerr << "Error: Bad image size." << std::endl;
    return -1;
  }

  int64_t rows = 0, cols = 0;
  clIm2Col.ComputeMatrixDimensions(rows, cols, a_i64ImageSize);

  std::vector<int64_t> vIndexMatrix(rows*cols, 0);

  clIm2Col.ExtractIndexMatrix(vIndexMatrix.data(), a_i64ImageSize);

  std::cout << "Index matrix: " << std::endl;

  for (int64_t i = 0; i < rows; ++i) {
    for (int64_t j = 0; j < cols; ++j)
      std::cout << vIndexMatrix[cols*i + j] << ' ';

    std::cout << std::endl;
  }

  int64_t *d_indexMatrix = nullptr;
  if (hipMalloc((void **)(&d_indexMatrix), vIndexMatrix.size()*sizeof(int64_t)) != hipSuccess) {
    std::cerr << "Error: hipMalloc failed." << std::endl;
    return -1;
  }

  clIm2Col.ExtractIndexMatrixGPU(d_indexMatrix, a_i64ImageSize);

  std::vector<int64_t> vIndexMatrix2(vIndexMatrix.size());

  if (hipMemcpy(vIndexMatrix2.data(), d_indexMatrix, vIndexMatrix2.size()*sizeof(int64_t), hipMemcpyDeviceToHost) != hipSuccess) {
    std::cerr << "Error: hipMemcpy failed." << std::endl;
    return -1;
  }

  std::cout << "GPU index matrix: " << std::endl;

  for (int64_t i = 0; i < rows; ++i) {
    for (int64_t j = 0; j < cols; ++j)
      std::cout << vIndexMatrix2[cols*i + j] << ' ';

    std::cout << std::endl;
  }

  if (std::equal(vIndexMatrix.begin(), vIndexMatrix.end(), vIndexMatrix2.begin())) {
    std::cout << "Info: Good." << std::endl;
  }
  else {
    std::cerr << "Error: Not equal." << std::endl;
  }

  return 0;
}
#endif 

