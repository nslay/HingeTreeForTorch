#include "hip/hip_runtime.h"
/*-
 * Nathan Lay
 * AI Resource at National Cancer Institute
 * National Institutes of Health
 * November 2020
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR(S) ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR(S) BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <algorithm>
#include <numeric>
#include <functional>

#include "torch/extension.h"
#include "HingeTreeCommon.cuh"

#include <hip/hip_runtime.h>

// From: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
// And from: https://stackoverflow.com/questions/39274472/error-function-atomicadddouble-double-has-already-been-defined
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

//#if __CUDA_ARCH__ < 600
#else
static inline __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

namespace {

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void ForwardKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, const RealType *d_inWeights, RealType *d_outData, 
    int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const KeyType key = keyMarginTuple.leafKey;
    const RealType signedMargin = keyMarginTuple.signedMargin;
    const RealType margin = std::abs(signedMargin);

    const RealType * const d_leafWeights = d_inWeights + (j*i64WeightsStride + key)*i64InnerWeightsNum;
    RealType * const d_out = d_outData + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;

    for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
      d_out[l] = d_leafWeights[l] * margin;
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void ReachabilityKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, int64_t *d_outCounts, 
    int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64NumTrees, int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t j = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (j < i64NumTrees) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;
    int64_t * const d_counts = d_outCounts + j*i64WeightsStride;

    for (int64_t i = 0; i < i64OuterNum; ++i) {
      for (int64_t k = 0; k < i64InnerDataNum; ++k) {
        const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

        // leaf key, margin, ordinal index
        const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

        const KeyType key = keyMarginTuple.leafKey;
        d_counts[key] += 1;
      }
    }
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void LeafMapKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, int64_t *d_outData, 
    int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64NumTrees, int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const KeyType key = keyMarginTuple.leafKey;

    d_outData[(i*i64NumTrees + j)*i64InnerDataNum + k] = key;
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void MarginMapKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, RealType *d_outMargins, 
    int64_t *d_outOrdinals, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64NumTrees, int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const RealType signedMargin = keyMarginTuple.signedMargin;
    const KeyType thresholdIndex = keyMarginTuple.thresholdIndex;

    d_outMargins[(i*i64NumTrees + j)*i64InnerDataNum + k] = signedMargin;
    d_outOrdinals[(i*i64NumTrees + j)*i64InnerDataNum + k] = d_ordinals[thresholdIndex];
  }
}

// Execute each example on one tree per thread for deterministic gradients
// This is potentially *really* slow
template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void DeterministicBackwardThresholdsKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, const RealType *d_inWeights, 
    const RealType *d_outDataGradient, RealType *d_inThresholdsGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  // Tree index
  const int64_t j = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (j < i64NumTrees) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;
    RealType * const d_thresholdsGradient = d_inThresholdsGradient + j*i64ThresholdStride;

    for (int64_t i = 0; i < i64OuterNum; ++i) {
      for (int64_t k = 0; k < i64InnerDataNum; ++k) {
        const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

        const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

        const KeyType key = keyMarginTuple.leafKey;
        const RealType signedMargin = keyMarginTuple.signedMargin;
        const KeyType thresholdIndex = keyMarginTuple.thresholdIndex;

        const RealType sign = RealType((RealType(0) < signedMargin) - (signedMargin < RealType(0)));

        const RealType * const d_leafWeights = d_inWeights + (j*i64WeightsStride + key)*i64InnerWeightsNum;
        const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;

        RealType tmpSum = RealType(0);
        for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
          tmpSum += d_leafWeights[l] * d_outGradient[l];

        tmpSum *= -sign;

        d_thresholdsGradient[thresholdIndex] += tmpSum; // Do this just once
      }
    }
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void BackwardThresholdsKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, const RealType *d_inWeights, 
    const RealType *d_outDataGradient, RealType *d_inThresholdsGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;
    RealType * const d_thresholdsGradient = d_inThresholdsGradient + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const KeyType key = keyMarginTuple.leafKey;
    const RealType signedMargin = keyMarginTuple.signedMargin;
    const KeyType thresholdIndex = keyMarginTuple.thresholdIndex;

    const RealType sign = RealType((RealType(0) < signedMargin) - (signedMargin < RealType(0)));

    const RealType * const d_leafWeights = d_inWeights + (j*i64WeightsStride + key)*i64InnerWeightsNum;
    const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;

    RealType tmpSum = RealType(0);
    for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
      tmpSum += d_leafWeights[l] * d_outGradient[l];

    tmpSum *= -sign;

    atomicAdd(d_thresholdsGradient + thresholdIndex, tmpSum); // Do this just once

    //for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
      //d_thresholdsGradient[thresholdIndex] += -sign * d_leafWeights[l] * d_outGradient[l];
  }
}

// Execute each example on one tree per thread for deterministic gradients
// This is potentially *really* slow
template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void DeterministicBackwardWeightsKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, /*const RealType *d_inWeights,*/
    const RealType *d_outDataGradient, RealType *d_inWeightsGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  // Tree index
  const int64_t j = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (j < i64NumTrees) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    for (int64_t i = 0; i < i64OuterNum; ++i) {
      for (int64_t k = 0; k < i64InnerDataNum; ++k) {
        const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

        // leaf key, margin, ordinal index
        const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

        const KeyType key = keyMarginTuple.leafKey;
        const RealType signedMargin = keyMarginTuple.signedMargin;
        const RealType margin = std::abs(signedMargin);

        const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;
        RealType * const d_leafWeightsGradient = d_inWeightsGradient + (j*i64WeightsStride + key)*i64InnerWeightsNum;

        for (int64_t l = 0; l < i64InnerWeightsNum; ++l) {
          d_leafWeightsGradient[l] += margin * d_outGradient[l]; 
        }
      }
    }
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void BackwardWeightsKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, /*const RealType *d_inWeights,*/
    const RealType *d_outDataGradient, RealType *d_inWeightsGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const KeyType key = keyMarginTuple.leafKey;
    const RealType signedMargin = keyMarginTuple.signedMargin;
    const RealType margin = std::abs(signedMargin);

    const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;
    RealType * const d_leafWeightsGradient = d_inWeightsGradient + (j*i64WeightsStride + key)*i64InnerWeightsNum;

    for (int64_t l = 0; l < i64InnerWeightsNum; ++l) {
      atomicAdd(d_leafWeightsGradient + l, margin * d_outGradient[l]); // Really bad!
      //d_leafWeightsGradient[l] += margin * d_outGradient[l];
    }
  }
}
// Execute all trees on one example per thread for deterministic gradients
// This is potentially *really* slow
template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void DeterministicBackwardDataKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, const RealType *d_inWeights, 
    const RealType *d_outDataGradient, RealType *d_inDataGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  // Batch and inner indices
  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && k < i64InnerDataNum) {
    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    for (int64_t j = 0; j < i64NumTrees; ++j) {
      const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
      const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

      // leaf key, margin, ordinal index
      const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

      const KeyType key = keyMarginTuple.leafKey;
      const RealType signedMargin = keyMarginTuple.signedMargin;
      const KeyType thresholdIndex = keyMarginTuple.thresholdIndex;
      const int64_t i64InputIndex = d_ordinals[thresholdIndex];

      const RealType * const d_leafWeights = d_inWeights + (j*i64WeightsStride + key)*i64InnerWeightsNum;
      const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;

      const RealType sign = RealType((RealType(0) < signedMargin) - (signedMargin < RealType(0)));
      RealType tmpSum = RealType(0);

      for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
        tmpSum += d_leafWeights[l] * d_outGradient[l];

      tmpSum *= sign;

      d_inDataGradient[(i*i64NumChannels + i64InputIndex)*i64InnerDataNum + k] += tmpSum; 
    }
  }
}

template<typename TreeTraitsTypeGPU, typename RealType>
__global__ void BackwardDataKernel(const RealType *d_inData, const RealType *d_inThresholds, const int64_t *d_inOrdinals, const RealType *d_inWeights, 
    const RealType *d_outDataGradient, RealType *d_inDataGradient, int64_t i64TreeDepth, int64_t i64ThresholdStride, int64_t i64WeightsStride, int64_t i64InnerWeightsNum, int64_t i64NumTrees, 
    int64_t i64OuterNum, int64_t i64NumChannels, int64_t i64InnerDataNum) {

  typedef typename TreeTraitsTypeGPU::KeyType KeyType;

  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i64OuterNum && j < i64NumTrees && k < i64InnerDataNum) {
    const RealType * const d_thresholds = d_inThresholds + j*i64ThresholdStride;
    const int64_t * const d_ordinals = d_inOrdinals + j*i64ThresholdStride;

    const RealType * const d_row = d_inData + ((i*i64NumChannels + 0)*i64InnerDataNum + k);

    // leaf key, margin, ordinal index
    const auto keyMarginTuple = TreeTraitsTypeGPU::ComputeKeyAndSignedMargin(d_row, d_thresholds, d_ordinals, i64TreeDepth, i64InnerDataNum);

    const KeyType key = keyMarginTuple.leafKey;
    const RealType signedMargin = keyMarginTuple.signedMargin;
    const KeyType thresholdIndex = keyMarginTuple.thresholdIndex;
    const int64_t i64InputIndex = d_ordinals[thresholdIndex];

    const RealType * const d_leafWeights = d_inWeights + (j*i64WeightsStride + key)*i64InnerWeightsNum;
    const RealType * const d_outGradient = d_outDataGradient + ((i*i64NumTrees + j)*i64InnerDataNum + k)*i64InnerWeightsNum;

    const RealType sign = RealType((RealType(0) < signedMargin) - (signedMargin < RealType(0)));
    RealType tmpSum = RealType(0);

    for (int64_t l = 0; l < i64InnerWeightsNum; ++l)
      tmpSum += d_leafWeights[l] * d_outGradient[l];

    tmpSum *= sign;

    atomicAdd(d_inDataGradient + ((i*i64NumChannels + i64InputIndex)*i64InnerDataNum + k), tmpSum); // Do this just once

    //d_inDataGradient[(i*iNumChannels + iInputIndex)*iInnerDataNum + k] += tmpSum; // Do this just once
  }
}

} // end anonymous namespace

typedef c10::IntArrayRef IntArrayRef;

template<typename RealType, typename TreeTraitsType>
torch::Tensor hingetree_gpu_forward(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return torch::Tensor();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return torch::Tensor();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return torch::Tensor();
 
  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  const RealType * const p_inWeights = inWeights.data_ptr<RealType>();
  
  std::vector<IntArrayRef::value_type> vSizes;
  
  vSizes.resize(2);
  vSizes[0] = inData.sizes()[0]; // batch size
  vSizes[1] = inWeights.sizes()[0]; // Number of trees
  
  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    vSizes.insert(vSizes.end(), inDataSlice.begin(), inDataSlice.end());
  }

  if (inWeights.sizes().size() > 2) {
    auto inWeightsSlice = inWeights.sizes().slice(2);
    vSizes.insert(vSizes.end(), inWeightsSlice.begin(), inWeightsSlice.end());
  }
  
  torch::Tensor outData = torch::empty(IntArrayRef(vSizes.data(), vSizes.size()), clOptions);
  
  RealType * const p_outData = outData.data_ptr<RealType>();
  
  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
  }
  
  int64_t i64InnerWeightsNum = 1;
  
  {
    auto inWeightsSlice = inWeights.sizes().slice(2);
    i64InnerWeightsNum = std::accumulate(inWeightsSlice.begin(), inWeightsSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
  }
  
  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64InnerDataNum + threadsPerBlock.x-1)/threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1)/threadsPerBlock.y, (i64NumTrees + threadsPerBlock.z-1)/threadsPerBlock.z);

  ForwardKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_inWeights, p_outData, 
    i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

  return outData;
}

template<typename RealType, typename TreeTraitsType>
std::vector<torch::Tensor> hingetree_gpu_backward_deterministic(torch::Tensor inData, bool bInDataGrad, torch::Tensor inThresholds, bool bInThresholdsGrad, torch::Tensor inOrdinals, bool bInOrdinalsGrad, torch::Tensor inWeights, bool bInWeightsGrad, torch::Tensor outDataGrad) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (bInOrdinalsGrad) // Not differentiable, ever!
    return std::vector<torch::Tensor>();
  
  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return std::vector<torch::Tensor>();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return std::vector<torch::Tensor>();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return std::vector<torch::Tensor>();
  
  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return std::vector<torch::Tensor>();

  std::vector<IntArrayRef::value_type> vSizes;
  
  vSizes.resize(2);
  vSizes[0] = inData.sizes()[0]; // batch size
  vSizes[1] = inWeights.sizes()[0]; // Number of trees

  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
    vSizes.insert(vSizes.end(), inDataSlice.begin(), inDataSlice.end());
  }
  
  int64_t i64InnerWeightsNum = 1;
  
  {
    auto inWeightsSlice = inWeights.sizes().slice(2);
    i64InnerWeightsNum = std::accumulate(inWeightsSlice.begin(), inWeightsSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
    vSizes.insert(vSizes.end(), inWeightsSlice.begin(), inWeightsSlice.end());
  }

  // Sanity check on outDataGrad
  if (outDataGrad.sizes() != IntArrayRef(vSizes.data(), vSizes.size()))
    return std::vector<torch::Tensor>();

  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  const RealType * const p_inWeights = inWeights.data_ptr<RealType>();
  const RealType * const p_outDataGrad = outDataGrad.data_ptr<RealType>();

  std::vector<torch::Tensor> vGradTensors(4);

  if (bInDataGrad) {
    torch::Tensor inDataGrad = torch::zeros_like(inData);
    RealType * const p_inDataGrad = inDataGrad.data_ptr<RealType>();

    const dim3 threadsPerBlock(32,32);
    const dim3 numBlocks((i64InnerDataNum + threadsPerBlock.x-1)/threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1)/threadsPerBlock.y);

    DeterministicBackwardDataKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_inWeights, p_outDataGrad, p_inDataGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[0] = inDataGrad;
  }
  
  if (bInThresholdsGrad) {
    torch::Tensor inThresholdsGrad = torch::zeros_like(inThresholds);
    RealType * const p_inThresholdsGrad = inThresholdsGrad.data_ptr<RealType>();
    
    const dim3 threadsPerBlock(1024);
    const dim3 numBlocks((i64NumTrees + threadsPerBlock.x-1)/threadsPerBlock.x);

    DeterministicBackwardThresholdsKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_inWeights, p_outDataGrad, p_inThresholdsGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[1] = inThresholdsGrad;
  }
  
  if (bInWeightsGrad) {
    torch::Tensor inWeightsGrad = torch::zeros_like(inWeights);
    RealType * const p_inWeightsGrad = inWeightsGrad.data_ptr<RealType>();
    
    const dim3 threadsPerBlock(1024);
    const dim3 numBlocks((i64NumTrees + threadsPerBlock.x-1)/threadsPerBlock.x);

    DeterministicBackwardWeightsKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_outDataGrad, p_inWeightsGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[3] = inWeightsGrad;
  }

  return vGradTensors;
}

template<typename RealType, typename TreeTraitsType>
std::vector<torch::Tensor> hingetree_gpu_backward(torch::Tensor inData, bool bInDataGrad, torch::Tensor inThresholds, bool bInThresholdsGrad, torch::Tensor inOrdinals, bool bInOrdinalsGrad, torch::Tensor inWeights, bool bInWeightsGrad, torch::Tensor outDataGrad) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (bInOrdinalsGrad) // Not differentiable, ever!
    return std::vector<torch::Tensor>();
  
  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return std::vector<torch::Tensor>();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return std::vector<torch::Tensor>();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return std::vector<torch::Tensor>();
  
  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return std::vector<torch::Tensor>();

  std::vector<IntArrayRef::value_type> vSizes;
  
  vSizes.resize(2);
  vSizes[0] = inData.sizes()[0]; // batch size
  vSizes[1] = inWeights.sizes()[0]; // Number of trees

  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
    vSizes.insert(vSizes.end(), inDataSlice.begin(), inDataSlice.end());
  }
  
  int64_t i64InnerWeightsNum = 1;
  
  {
    auto inWeightsSlice = inWeights.sizes().slice(2);
    i64InnerWeightsNum = std::accumulate(inWeightsSlice.begin(), inWeightsSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
    vSizes.insert(vSizes.end(), inWeightsSlice.begin(), inWeightsSlice.end());
  }

  // Sanity check on outDataGrad
  if (outDataGrad.sizes() != IntArrayRef(vSizes.data(), vSizes.size()))
    return std::vector<torch::Tensor>();

  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  const RealType * const p_inWeights = inWeights.data_ptr<RealType>();
  const RealType * const p_outDataGrad = outDataGrad.data_ptr<RealType>();

  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64InnerDataNum + threadsPerBlock.x-1)/threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1)/threadsPerBlock.y, (i64NumTrees + threadsPerBlock.z-1)/threadsPerBlock.z);

  std::vector<torch::Tensor> vGradTensors(4);

  if (bInDataGrad) {
    torch::Tensor inDataGrad = torch::zeros_like(inData);
    RealType * const p_inDataGrad = inDataGrad.data_ptr<RealType>();
    
    BackwardDataKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_inWeights, p_outDataGrad, p_inDataGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[0] = inDataGrad;
  }
  
  if (bInThresholdsGrad) {
    torch::Tensor inThresholdsGrad = torch::zeros_like(inThresholds);
    RealType * const p_inThresholdsGrad = inThresholdsGrad.data_ptr<RealType>();
    
    BackwardThresholdsKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_inWeights, p_outDataGrad, p_inThresholdsGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[1] = inThresholdsGrad;
  }
  
  if (bInWeightsGrad) {
    torch::Tensor inWeightsGrad = torch::zeros_like(inWeights);
    RealType * const p_inWeightsGrad = inWeightsGrad.data_ptr<RealType>();
    
    BackwardWeightsKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_outDataGrad, p_inWeightsGrad, 
      i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64InnerWeightsNum, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

    vGradTensors[3] = inWeightsGrad;
  }

  return vGradTensors;
}

template<typename RealType, typename TreeTraitsType>
torch::Tensor hingetree_gpu_reachability(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return torch::Tensor();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return torch::Tensor();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return torch::Tensor();
 
  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  
  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());
  torch::Tensor outCounts = torch::zeros(inWeights.sizes().slice(0,2), clOptions.dtype(torch::kInt64));
  
  int64_t * const p_outCounts = outCounts.data_ptr<int64_t>();
  
  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
  }
  
  const dim3 threadsPerBlock(1024);
  const dim3 numBlocks((i64NumTrees + threadsPerBlock.x-1)/threadsPerBlock.x);

  ReachabilityKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_outCounts,
    i64TreeDepth, i64NumDecisionsPerTree, i64NumLeavesPerTree, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

  return outCounts;
}

template<typename RealType, typename TreeTraitsType>
torch::Tensor hingetree_gpu_leafmap(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return torch::Tensor();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return torch::Tensor();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return torch::Tensor();
 
  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  
  std::vector<IntArrayRef::value_type> vSizes;
  
  vSizes.resize(2);
  vSizes[0] = inData.sizes()[0]; // batch size
  vSizes[1] = inWeights.sizes()[0]; // Number of trees
  
  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    vSizes.insert(vSizes.end(), inDataSlice.begin(), inDataSlice.end());
  }

  torch::Tensor outData = torch::empty(IntArrayRef(vSizes.data(), vSizes.size()), clOptions.dtype(torch::kInt64));
  
  int64_t * const p_outData = outData.data_ptr<int64_t>();
  
  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
  }
  
  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64InnerDataNum + threadsPerBlock.x-1)/threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1)/threadsPerBlock.y, (i64NumTrees + threadsPerBlock.z-1)/threadsPerBlock.z);

  LeafMapKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_outData, 
    i64TreeDepth, i64NumDecisionsPerTree, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

  return outData;
}

template<typename RealType, typename TreeTraitsType>
std::vector<torch::Tensor> hingetree_gpu_marginmap(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights) {
  typedef bleak::HingeTreeCommonGPU<TreeTraitsType> TreeTraitsTypeGPU;

  if (inData.dim() < 2 || inThresholds.dim() != 2 || inOrdinals.dim() != 2 || inWeights.dim() < 2)
    return std::vector<torch::Tensor>();

  if (inThresholds.sizes() != inOrdinals.sizes() || inWeights.sizes()[0] != inThresholds.sizes()[0])
    return std::vector<torch::Tensor>();
  
  const int64_t i64NumTrees = inWeights.sizes()[0];
  const int64_t i64NumLeavesPerTree = inWeights.sizes()[1];
  const int64_t i64TreeDepth = TreeTraitsType::ComputeDepth(i64NumLeavesPerTree);
  
  if (i64TreeDepth > TreeTraitsType::GetMaxDepth() || inThresholds.sizes()[1] != TreeTraitsType::GetThresholdCount(i64TreeDepth))
    return std::vector<torch::Tensor>();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64NumDecisionsPerTree = inThresholds.sizes()[1];

  if (inOrdinals.min().to(torch::kCPU).item<int64_t>() < 0 || inOrdinals.max().to(torch::kCPU).item<int64_t>() >= i64NumChannels)
    return std::vector<torch::Tensor>();
 
  const RealType * const p_inData = inData.data_ptr<RealType>();
  const RealType * const p_inThresholds = inThresholds.data_ptr<RealType>();
  const int64_t * const p_inOrdinals = inOrdinals.data_ptr<int64_t>();
  
  std::vector<IntArrayRef::value_type> vSizes;
  
  vSizes.resize(2);
  vSizes[0] = inData.sizes()[0]; // batch size
  vSizes[1] = inWeights.sizes()[0]; // Number of trees
  
  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    vSizes.insert(vSizes.end(), inDataSlice.begin(), inDataSlice.end());
  }

  torch::Tensor outMargins = torch::empty(IntArrayRef(vSizes.data(), vSizes.size()), clOptions);
  torch::Tensor outOrdinals = torch::empty(IntArrayRef(vSizes.data(), vSizes.size()), clOptions.dtype(torch::kInt64));
  
  RealType * const p_outMargins = outMargins.data_ptr<RealType>();
  int64_t * const p_outOrdinals = outOrdinals.data_ptr<int64_t>();
  
  int64_t i64InnerDataNum = 1;
  
  {
    auto inDataSlice = inData.sizes().slice(2);
    i64InnerDataNum = std::accumulate(inDataSlice.begin(), inDataSlice.end(), (int64_t)1, std::multiplies<IntArrayRef::value_type>());
  }
  
  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64InnerDataNum + threadsPerBlock.x-1)/threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1)/threadsPerBlock.y, (i64NumTrees + threadsPerBlock.z-1)/threadsPerBlock.z);

  MarginMapKernel<TreeTraitsTypeGPU><<<numBlocks, threadsPerBlock>>>(p_inData, p_inThresholds, p_inOrdinals, p_outMargins, p_outOrdinals, 
    i64TreeDepth, i64NumDecisionsPerTree, i64NumTrees, i64BatchSize, i64NumChannels, i64InnerDataNum);

  return { outMargins, outOrdinals };
}

template torch::Tensor hingetree_gpu_forward<float, bleak::HingeTreeCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_forward<double, bleak::HingeTreeCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template torch::Tensor hingetree_gpu_forward<float, bleak::HingeFernCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_forward<double, bleak::HingeFernCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template std::vector<torch::Tensor> hingetree_gpu_backward<float, bleak::HingeTreeCommon<float>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);
template std::vector<torch::Tensor> hingetree_gpu_backward<double, bleak::HingeTreeCommon<double>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);

template std::vector<torch::Tensor> hingetree_gpu_backward_deterministic<float, bleak::HingeTreeCommon<float>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);
template std::vector<torch::Tensor> hingetree_gpu_backward_deterministic<double, bleak::HingeTreeCommon<double>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);

template std::vector<torch::Tensor> hingetree_gpu_backward<float, bleak::HingeFernCommon<float>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);
template std::vector<torch::Tensor> hingetree_gpu_backward<double, bleak::HingeFernCommon<double>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);

template std::vector<torch::Tensor> hingetree_gpu_backward_deterministic<float, bleak::HingeFernCommon<float>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);
template std::vector<torch::Tensor> hingetree_gpu_backward_deterministic<double, bleak::HingeFernCommon<double>>(torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor, bool, torch::Tensor);

template torch::Tensor hingetree_gpu_reachability<float, bleak::HingeTreeCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_reachability<double, bleak::HingeTreeCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template torch::Tensor hingetree_gpu_reachability<float, bleak::HingeFernCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_reachability<double, bleak::HingeFernCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template torch::Tensor hingetree_gpu_leafmap<float, bleak::HingeTreeCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_leafmap<double, bleak::HingeTreeCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template torch::Tensor hingetree_gpu_leafmap<float, bleak::HingeFernCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template torch::Tensor hingetree_gpu_leafmap<double, bleak::HingeFernCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template std::vector<torch::Tensor> hingetree_gpu_marginmap<float, bleak::HingeTreeCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template std::vector<torch::Tensor> hingetree_gpu_marginmap<double, bleak::HingeTreeCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

template std::vector<torch::Tensor> hingetree_gpu_marginmap<float, bleak::HingeFernCommon<float>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);
template std::vector<torch::Tensor> hingetree_gpu_marginmap<double, bleak::HingeFernCommon<double>>(torch::Tensor inData, torch::Tensor inThresholds, torch::Tensor inOrdinals, torch::Tensor inWeights);

