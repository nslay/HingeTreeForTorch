#include "hip/hip_runtime.h"
/*-
 * Nathan Lay
 * AI Resource at National Cancer Institute
 * National Institutes of Health
 * May 2023
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR(S) ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR(S) BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "torch/extension.h"

typedef c10::IntArrayRef IntArrayRef;

// From: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html
// And from: https://stackoverflow.com/questions/39274472/error-function-atomicadddouble-double-has-already-been-defined
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

//#if __CUDA_ARCH__ < 600
#else
static inline __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

namespace {

// Make struct that can be conveniently passed to kernels by value
template<unsigned int Dimension>
struct Size {
  constexpr static size_t size = Dimension;
  int64_t data[Dimension];
};

__device__ void contract2d_coords(int64_t &xo, int64_t &yo, int64_t &xw, int64_t &yw, int64_t k, const int64_t sizes[4]) {
  int64_t q = k / sizes[3];
  xw = k - q*sizes[3];

  k = q;
  q = k / sizes[2];

  yw = k - q*sizes[2];

  k = q;
  q = k / sizes[1];

  xo = k - q*sizes[1];

  k = q;
  q = k / sizes[0];

  yo = k - q*sizes[0];
}

__device__ void contract3d_coords(int64_t &xo, int64_t &yo, int64_t &zo, int64_t &xw, int64_t &yw, int64_t &zw, int64_t k, const int64_t sizes[6]) {
  int64_t q = k / sizes[5];
  xw = k - q*sizes[5];

  k = q;
  q = k / sizes[4];

  yw = k - q*sizes[4];

  k = q;
  q = k / sizes[3];

  zw = k - q*sizes[3];

  k = q;
  q = k / sizes[2];

  xo = k - q*sizes[2];

  k = q;
  q = k / sizes[1];

  yo = k - q*sizes[1];

  k = q;
  q = k / sizes[0];

  zo = k - q*sizes[0];
}

template<typename RealType>
__global__ void contract2d_kernel(RealType *d_outData, const RealType *d_inData, Size<6> stOutSize, Size<2> stInSize, Size<2> stPadding) {
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;

  const int64_t i64Size = stOutSize.data[2]*stOutSize.data[3]*stOutSize.data[4]*stOutSize.data[5];

  if (i < stOutSize.data[0] && j < stOutSize.data[1] && k < i64Size) {
    int64_t xo=0, yo=0, xw=0, yw=0;

    contract2d_coords(xo, yo, xw, yw, k, stOutSize.data + 2);

    const int64_t yi = yo*stOutSize.data[4] - stPadding.data[0] + yw;
    const int64_t xi = xo*stOutSize.data[5] - stPadding.data[1] + xw;

    if (yi >= 0 && xi >= 0 && yi < stInSize.data[0] && xi < stInSize.data[1])
      d_outData[(i*stOutSize.data[1] + j)*i64Size + k] = d_inData[((i*stOutSize.data[1] + j)*stInSize.data[0] + yi)*stInSize.data[1] + xi];
  }
}

template<typename RealType>
__global__ void contract3d_kernel(RealType *d_outData, const RealType *d_inData, Size<8> stOutSize, Size<3> stInSize, Size<3> stPadding) {
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;

  const int64_t i64Size = stOutSize.data[2]*stOutSize.data[3]*stOutSize.data[4]*stOutSize.data[5]*stOutSize.data[6]*stOutSize.data[7];

  if (i < stOutSize.data[0] && j < stOutSize.data[1] && k < i64Size) {
    int64_t xo=0, yo=0, zo=0, xw=0, yw=0, zw=0;

    contract3d_coords(xo, yo, zo, xw, yw, zw, k, stOutSize.data + 2);

    const int64_t zi = zo*stOutSize.data[5] - stPadding.data[0] + zw;
    const int64_t yi = yo*stOutSize.data[6] - stPadding.data[1] + yw;
    const int64_t xi = xo*stOutSize.data[7] - stPadding.data[2] + xw;

    if (zi >= 0 && yi >= 0 && xi >= 0 && zi < stInSize.data[0] && yi < stInSize.data[1] && xi < stInSize.data[2])
      d_outData[(i*stOutSize.data[1] + j)*i64Size + k] = d_inData[(((i*stOutSize.data[1] + j)*stInSize.data[0] + zi)*stInSize.data[1] + yi)*stInSize.data[2] + xi];
  }
}

template<typename RealType>
__global__ void expand2d_kernel(RealType *d_outData, const RealType *d_inData, Size<4> stOutSize, Size<4> stInSize, Size<2> stPadding) {
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;

  const int64_t i64Size = stInSize.data[0]*stInSize.data[1]*stInSize.data[2]*stInSize.data[3];

  if (i < stOutSize.data[0] && j < stOutSize.data[1] && k < i64Size) {
    int64_t xi=0, yi=0, xw=0, yw=0;
    contract2d_coords(xi, yi, xw, yw, k, stInSize.data);

    const int64_t yo = yi*stInSize.data[2] - stPadding.data[0] + yw;
    const int64_t xo = xi*stInSize.data[3] - stPadding.data[1] + xw;

    if (yo >= 0 && xo >= 0 && yo < stOutSize.data[2] && xo < stOutSize.data[3])
      d_outData[((i*stOutSize.data[1] + j)*stOutSize.data[2] + yo)*stOutSize.data[3] + xo] = d_inData[(i*stOutSize.data[1] + j)*i64Size + k];
    
  }
}

template<typename RealType>
__global__ void expand3d_kernel(RealType *d_outData, const RealType *d_inData, Size<5> stOutSize, Size<6> stInSize, Size<3> stPadding) {
  const int64_t k = (int64_t)blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t i = (int64_t)blockIdx.y * blockDim.y + threadIdx.y;
  const int64_t j = (int64_t)blockIdx.z * blockDim.z + threadIdx.z;

  const int64_t i64Size = stInSize.data[0]*stInSize.data[1]*stInSize.data[2]*stInSize.data[3]*stInSize.data[4]*stInSize.data[5];

  if (i < stOutSize.data[0] && j < stOutSize.data[1] && k < i64Size) {
    int64_t xi=0, yi=0, zi=0, xw=0, yw=0, zw=0;
    contract3d_coords(xi, yi, zi, xw, yw, zw, k, stInSize.data);

    const int64_t zo = zi*stInSize.data[3] - stPadding.data[0] + zw;
    const int64_t yo = yi*stInSize.data[4] - stPadding.data[1] + yw;
    const int64_t xo = xi*stInSize.data[5] - stPadding.data[2] + xw;

    if (zo >= 0 && yo >= 0 && xo >= 0 && zo < stOutSize.data[2] && yo < stOutSize.data[3] && xo < stOutSize.data[4])
      d_outData[(((i*stOutSize.data[1] + j)*stOutSize.data[2] + zo)*stOutSize.data[3] + yo)*stOutSize.data[4] + xo] = d_inData[(i*stOutSize.data[1] + j)*i64Size + k];
    
  }
}

} // end anonymous namespace

template<typename RealType>
torch::Tensor contract2d_gpu(torch::Tensor inData, const int64_t a_i64Window[2], const int64_t a_i64Padding[2]) {
  if (inData.dim() != 4 || a_i64Padding[0] < 0 || a_i64Padding[1] < 0)
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64Height = inData.sizes()[2];
  const int64_t i64Width = inData.sizes()[3];

  if (a_i64Window[0] < 1 || a_i64Window[1] < 1 || a_i64Window[0] > i64Height + 2*a_i64Padding[0] || a_i64Window[1] > i64Width + 2*a_i64Padding[1])
    return torch::Tensor();

  Size<6> stOutSize;
  stOutSize.data[0] = inData.sizes()[0];
  stOutSize.data[1] = inData.sizes()[1];
  stOutSize.data[2] = (i64Height + 2*a_i64Padding[0] - a_i64Window[0])/a_i64Window[0] + 1;
  stOutSize.data[3] = (i64Width + 2*a_i64Padding[1] - a_i64Window[1])/a_i64Window[1] + 1;
  stOutSize.data[4] = a_i64Window[0];
  stOutSize.data[5] = a_i64Window[1];

  const int64_t i64Size =  stOutSize.data[2]* stOutSize.data[3]* stOutSize.data[4]* stOutSize.data[5];

  Size<2> stPadding;
  stPadding.data[0] = a_i64Padding[0];
  stPadding.data[1] = a_i64Padding[1];

  Size<2> stInSize;
  stInSize.data[0] = i64Height;
  stInSize.data[1] = i64Width;

  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());

  torch::Tensor outData = torch::zeros(IntArrayRef(stOutSize.data, stOutSize.size), clOptions);

  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64Size + threadsPerBlock.x-1) / threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1) / threadsPerBlock.y, (i64NumChannels + threadsPerBlock.z-1) / threadsPerBlock.z);

  contract2d_kernel<<<numBlocks, threadsPerBlock>>>(outData.data_ptr<RealType>(), inData.data_ptr<RealType>(), stOutSize, stInSize, stPadding);

  return outData;
}

template<typename RealType>
torch::Tensor contract3d_gpu(torch::Tensor inData, const int64_t a_i64Window[3], const int64_t a_i64Padding[3]) {
  if (inData.dim() != 5 || a_i64Padding[0] < 0 || a_i64Padding[1] < 0 || a_i64Padding[2] < 0)
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t i64Depth = inData.sizes()[2];
  const int64_t i64Height = inData.sizes()[3];
  const int64_t i64Width = inData.sizes()[4];

  if (a_i64Window[0] < 1 || a_i64Window[1] < 1 || a_i64Window[2] < 1 || a_i64Window[0] > i64Depth + 2*a_i64Padding[0] || a_i64Window[1] > i64Height + 2*a_i64Padding[1] || a_i64Window[2] > i64Width + 2*a_i64Padding[2])
    return torch::Tensor();

  Size<8> stOutSize;
  stOutSize.data[0] = inData.sizes()[0];
  stOutSize.data[1] = inData.sizes()[1];
  stOutSize.data[2] = (i64Depth + 2*a_i64Padding[0] - a_i64Window[0])/a_i64Window[0] + 1;
  stOutSize.data[3] = (i64Height + 2*a_i64Padding[1] - a_i64Window[1])/a_i64Window[1] + 1;
  stOutSize.data[4] = (i64Width + 2*a_i64Padding[2] - a_i64Window[2])/a_i64Window[2] + 1;
  stOutSize.data[5] = a_i64Window[0];
  stOutSize.data[6] = a_i64Window[1];
  stOutSize.data[7] = a_i64Window[2];

  const int64_t i64Size =  stOutSize.data[2]* stOutSize.data[3]* stOutSize.data[4]* stOutSize.data[5] * stOutSize.data[6] * stOutSize.data[7];

  Size<3> stPadding;
  stPadding.data[0] = a_i64Padding[0];
  stPadding.data[1] = a_i64Padding[1];
  stPadding.data[2] = a_i64Padding[2];

  Size<3> stInSize;
  stInSize.data[0] = i64Depth;
  stInSize.data[1] = i64Height;
  stInSize.data[2] = i64Width;

  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());

  torch::Tensor outData = torch::zeros(IntArrayRef(stOutSize.data, stOutSize.size), clOptions);

  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64Size + threadsPerBlock.x-1) / threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1) / threadsPerBlock.y, (i64NumChannels + threadsPerBlock.z-1) / threadsPerBlock.z);

  contract3d_kernel<<<numBlocks, threadsPerBlock>>>(outData.data_ptr<RealType>(), inData.data_ptr<RealType>(), stOutSize, stInSize, stPadding);

  return outData;
}

template<typename RealType>
torch::Tensor expand2d_gpu(torch::Tensor inData, const int64_t a_i64Padding[2]) {
  if (inData.dim() != 6 || a_i64Padding[0] < 0 || a_i64Padding[1] < 0)
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t a_i64Window[2] = { inData.sizes()[4], inData.sizes()[5] };
  const int64_t i64Height = inData.sizes()[2]*a_i64Window[0] - ((2*a_i64Padding[0])/a_i64Window[0])*a_i64Window[0];
  const int64_t i64Width = inData.sizes()[3]*a_i64Window[1] - ((2*a_i64Padding[1])/a_i64Window[1])*a_i64Window[1];

  if (i64Height < 1 || i64Width < 1 )
    return torch::Tensor();
 
  Size<4> stInSize;
  stInSize.data[0] = inData.sizes()[2];
  stInSize.data[1] = inData.sizes()[3];
  stInSize.data[2] = inData.sizes()[4];
  stInSize.data[3] = inData.sizes()[5];

  const int64_t i64Size =  stInSize.data[0]* stInSize.data[1] * stInSize.data[2]* stInSize.data[3];

  Size<2> stPadding;
  stPadding.data[0] = a_i64Padding[0];
  stPadding.data[1] = a_i64Padding[1];

  Size<4> stOutSize;
  stOutSize.data[0] = inData.sizes()[0];   
  stOutSize.data[1] = inData.sizes()[1];
  stOutSize.data[2] = i64Height;
  stOutSize.data[3] = i64Width;

  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());

  torch::Tensor outData = torch::zeros(IntArrayRef(stOutSize.data, stOutSize.size), clOptions);

  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64Size + threadsPerBlock.x-1) / threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1) / threadsPerBlock.y, (i64NumChannels + threadsPerBlock.z-1) / threadsPerBlock.z);

  expand2d_kernel<<<numBlocks, threadsPerBlock>>>(outData.data_ptr<RealType>(), inData.data_ptr<RealType>(), stOutSize, stInSize, stPadding);

  return outData;
}

template<typename RealType>
torch::Tensor expand3d_gpu(torch::Tensor inData, const int64_t a_i64Padding[2]) {
  if (inData.dim() != 8 || a_i64Padding[0] < 0 || a_i64Padding[1] < 0 || a_i64Padding[2] < 0)
    return torch::Tensor();

  const int64_t i64BatchSize = inData.sizes()[0];
  const int64_t i64NumChannels = inData.sizes()[1];
  const int64_t a_i64Window[3] = { inData.sizes()[5], inData.sizes()[6], inData.sizes()[7] };
  const int64_t i64Depth = inData.sizes()[2]*a_i64Window[0] - ((2*a_i64Padding[0])/a_i64Window[0])*a_i64Window[0];
  const int64_t i64Height = inData.sizes()[3]*a_i64Window[1] - ((2*a_i64Padding[1])/a_i64Window[1])*a_i64Window[1];
  const int64_t i64Width = inData.sizes()[4]*a_i64Window[2] - ((2*a_i64Padding[2])/a_i64Window[2])*a_i64Window[2];

  if (i64Depth < 1 || i64Height < 1 || i64Width < 1)
    return torch::Tensor();

  Size<6> stInSize;
  stInSize.data[0] = inData.sizes()[2];
  stInSize.data[1] = inData.sizes()[3];
  stInSize.data[2] = inData.sizes()[4];
  stInSize.data[3] = inData.sizes()[5];
  stInSize.data[4] = inData.sizes()[6];
  stInSize.data[5] = inData.sizes()[7];

  const int64_t i64Size =  stInSize.data[0]* stInSize.data[1] * stInSize.data[2]* stInSize.data[3] * stInSize.data[4] * stInSize.data[5];

  Size<3> stPadding;
  stPadding.data[0] = a_i64Padding[0];
  stPadding.data[1] = a_i64Padding[1];
  stPadding.data[2] = a_i64Padding[2];

  Size<5> stOutSize;
  stOutSize.data[0] = inData.sizes()[0];   
  stOutSize.data[1] = inData.sizes()[1];
  stOutSize.data[2] = i64Depth;
  stOutSize.data[3] = i64Height;
  stOutSize.data[4] = i64Width;

  auto clOptions = torch::TensorOptions().dtype(inData.dtype()).device(inData.device());

  torch::Tensor outData = torch::zeros(IntArrayRef(stOutSize.data, stOutSize.size), clOptions);

  const dim3 threadsPerBlock(16,8,8);
  const dim3 numBlocks((i64Size + threadsPerBlock.x-1) / threadsPerBlock.x, (i64BatchSize + threadsPerBlock.y-1) / threadsPerBlock.y, (i64NumChannels + threadsPerBlock.z-1) / threadsPerBlock.z);

  expand3d_kernel<<<numBlocks, threadsPerBlock>>>(outData.data_ptr<RealType>(), inData.data_ptr<RealType>(), stOutSize, stInSize, stPadding);

  return outData;
}

template torch::Tensor contract2d_gpu<uint8_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<uint8_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<int8_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<int8_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<int16_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<int16_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<int32_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<int32_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<int64_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<int64_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<float>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<float>(torch::Tensor, const int64_t *);
template torch::Tensor contract2d_gpu<double>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand2d_gpu<double>(torch::Tensor, const int64_t *);

template torch::Tensor contract3d_gpu<uint8_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<uint8_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<int8_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<int8_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<int16_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<int16_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<int32_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<int32_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<int64_t>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<int64_t>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<float>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<float>(torch::Tensor, const int64_t *);
template torch::Tensor contract3d_gpu<double>(torch::Tensor, const int64_t *, const int64_t *);
template torch::Tensor expand3d_gpu<double>(torch::Tensor, const int64_t *);

